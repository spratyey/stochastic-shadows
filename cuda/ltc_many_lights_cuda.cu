#include "hip/hip_runtime.h"
#include "common.cuh"
#include "utils.cuh"

// These headers depend on functions included in common.cuh and utils.cuh
#include "ltc_many_lights_cuda.cuh"
#include "frostbite.cuh"

#include "ltc_utils.cuh"
#include "polygon_utils.cuh"
#include "lcg_random.h"

struct BST {
    int data = -1;
    int left = -1;
    int right = -1;
};

__device__ void stochasticTraverseLBVH(LightBVH* bvh, int bvhHeight, int rootNodeIdx, SurfaceInteraction& si, int& selectedIdx,
    float& lightSelectionPdf, vec2f randVec);
__device__ float deterministicTraverseLBVH(LightBVH* bvh, int bvhHeight, int rootNodeIdx, SurfaceInteraction& si, vec3f point, int& idx);

__device__ void selectFromLBVH(SurfaceInteraction& si, int& selectedIdx, float& lightSelectionPdf, vec2f rand0, vec2f rand1);
__device__ float pdfFromLBVH(SurfaceInteraction& si, vec3f point);

__device__ vec3f integrateOverPolygon(SurfaceInteraction& si, vec3f ltc_mat[3], vec3f ltc_mat_inv[3],
    float amplitude, vec3f iso_frame[3], TriLight& triLight);

__device__ vec3f estimateDirectLighting(SurfaceInteraction& si, LCGRand& rng, int type);
__device__ vec3f estimateDirectLightingLBVH(SurfaceInteraction& si, LCGRand& rng, int type);
__device__ vec3f ltcDirecLighingBaseline(SurfaceInteraction& si, LCGRand& rng);

__device__ vec3f ltcDirectLightingLBVH(SurfaceInteraction& si, LCGRand& rng);


__device__ vec3f sampleLightSource(SurfaceInteraction si, int lightIdx, float lightSelectionPdf, vec2f rand, bool mis);
__device__ vec3f sampleBRDF(SurfaceInteraction si, float lightSelectionPdf, vec2f rand, bool mis);

OPTIX_RAYGEN_PROGRAM(rayGen)()
{
    const RayGenData& self = owl::getProgramData<RayGenData>();
    const vec2i pixelId = owl::getLaunchIndex();
    const int fbOfs = pixelId.x + self.frameBufferSize.x * pixelId.y;

    LCGRand rng = get_rng(optixLaunchParams.accumId, make_uint2(pixelId.x, pixelId.y), 
        make_uint2(self.frameBufferSize.x, self.frameBufferSize.y));

    const vec2f screen = (vec2f(pixelId) + +vec2f(lcg_randomf(rng), lcg_randomf(rng))) / vec2f(self.frameBufferSize);
    RadianceRay ray;
    ray.origin
        = optixLaunchParams.camera.pos;
    ray.direction
        = normalize(optixLaunchParams.camera.dir_00
            + screen.u * optixLaunchParams.camera.dir_du
            + screen.v * optixLaunchParams.camera.dir_dv);

    SurfaceInteraction si;
    owl::traceRay(optixLaunchParams.world, ray, si);

    vec3f color(0.f, 0.f, 0.f);

    if (si.hit == false)
        color = si.diffuse;
    else if (optixLaunchParams.rendererType == DIFFUSE)
        color = si.diffuse;
    else if (optixLaunchParams.rendererType == ALPHA)
        color = si.alpha;
    else if (optixLaunchParams.rendererType == NORMALS)
        color = 0.5f * (si.n_geom + 1.f);
    // Direct lighting with MC
    else if (optixLaunchParams.rendererType == DIRECT_LIGHT_LSAMPLE) {
        if (si.isLight)
            color = si.emit;
        else
            color = estimateDirectLighting(si, rng, 0);
    }
    else if (optixLaunchParams.rendererType == DIRECT_LIGHT_BRDFSAMPLE) {
        if (si.isLight)
            color = si.emit;
        else
            color = estimateDirectLighting(si, rng, 1);
    }
    else if (optixLaunchParams.rendererType == DIRECT_LIGHT_MIS) {
        if (si.isLight)
            color = si.emit;
        else
            color = estimateDirectLighting(si, rng, 2);
    }
    // Direct lighting with MC and LBVH
    else if (optixLaunchParams.rendererType == DIRECT_LIGHT_LBVH_LSAMPLE) {
        if (si.isLight)
            color = si.emit;
        else
            color = estimateDirectLightingLBVH(si, rng, 0);
    }
    else if (optixLaunchParams.rendererType == DIRECT_LIGHT_LBVH_BRDFSAMPLE) {
        if (si.isLight)
            color = si.emit;
        else
            color = estimateDirectLightingLBVH(si, rng, 1);
    }
    else if (optixLaunchParams.rendererType == DIRECT_LIGHT_LBVH_MIS) {
        if (si.isLight)
            color = si.emit;
        else
            color = estimateDirectLightingLBVH(si, rng, 2);
    }
    // Direct lighting with LTC
    else if (optixLaunchParams.rendererType == LTC_BASELINE) {
        if (si.isLight)
            color = si.emit;
        else
            color = ltcDirecLighingBaseline(si, rng);
    }
    else if (optixLaunchParams.rendererType == LTC_LBVH_LINEAR) {
        if (si.isLight)
            color = si.emit;
        else
            color = ltcDirectLightingLBVH(si, rng);
    }
    else if (optixLaunchParams.rendererType == LTC_LBVH_BST) {
        if (si.isLight)
            color = si.emit;
        else
            color = ltcDirectLightingLBVH(si, rng);
    }

    if (optixLaunchParams.accumId > 0)
        color = color + vec3f(optixLaunchParams.accumBuffer[fbOfs]);

    optixLaunchParams.accumBuffer[fbOfs] = vec4f(color, 1.f);
    color = (1.f / (optixLaunchParams.accumId + 1)) * color;
    self.frameBuffer[fbOfs] = owl::make_rgba(color);   
}

OPTIX_CLOSEST_HIT_PROGRAM(triangleMeshCHShadow)()
{
    const TriangleMeshData& self = owl::getProgramData<TriangleMeshData>();
    const vec3i primitiveIndices = self.index[optixGetPrimitiveIndex()];
    ShadowRayData& srd = owl::getPRD<ShadowRayData>();

    if (self.isLight) {
        srd.visibility = vec3f(1.f);
        srd.point = barycentricInterpolate(self.vertex, primitiveIndices);
        srd.normal = normalize(barycentricInterpolate(self.normal, primitiveIndices));
        srd.emit = self.emit;

        vec3f v1 = self.vertex[primitiveIndices.x];
        vec3f v2 = self.vertex[primitiveIndices.y];
        vec3f v3 = self.vertex[primitiveIndices.z];
        srd.area = 0.5f * length(cross(v1 - v2, v3 - v2));

        srd.cg = (v1 + v2 + v3) / 3.f;
    }
    else {
        srd.visibility = vec3f(0.f);
    }

}

OPTIX_CLOSEST_HIT_PROGRAM(triangleMeshCH)()
{
    const TriangleMeshData& self = owl::getProgramData<TriangleMeshData>();
    const vec3i primitiveIndices = self.index[optixGetPrimitiveIndex()];

    SurfaceInteraction& si = owl::getPRD<SurfaceInteraction>();
    si.p = barycentricInterpolate(self.vertex, primitiveIndices);
    si.wo = owl::normalize( optixLaunchParams.camera.pos - si.p );
    si.uv = barycentricInterpolate(self.texCoord, primitiveIndices);
    si.n_geom = normalize( barycentricInterpolate(self.normal, primitiveIndices) );
    orthonormalBasis(si.n_geom, si.to_local, si.to_world);

    si.wo_local = normalize(apply_mat(si.to_local, si.wo));

    si.diffuse = self.diffuse;
    if (self.hasDiffuseTexture)
        si.diffuse = (vec3f) tex2D<float4>(self.diffuse_texture, si.uv.x, si.uv.y);

    si.alpha = self.alpha;
    if (self.hasAlphaTexture)
        si.alpha = tex2D<float4>(self.alpha_texture, si.uv.x, si.uv.y).x;
    si.alpha = clamp(si.alpha, 0.01f, 1.f);

    si.emit = self.emit;
    si.isLight = self.isLight;

    si.hit = true;
}

OPTIX_MISS_PROGRAM(miss)()
{
    const vec2i pixelId = owl::getLaunchIndex();
    const MissProgData& self = owl::getProgramData<MissProgData>();

    SurfaceInteraction& si = owl::getPRD<SurfaceInteraction>();
    si.hit = false;
    si.diffuse = self.const_color;
}

__device__
float deterministicTraverseLBVH(LightBVH* bvh, int bvhHeight, int rootNodeIdx, SurfaceInteraction& si, vec3f point, int& idx)
{
    float pdf = 1.f;

    int nodeIdx = rootNodeIdx;
    for (int i = 0; i < bvhHeight + 1; i++) {
        LightBVH node = bvh[nodeIdx];

        if (node.left == 0 && node.right == 0) {
            idx = node.primIdx;
            if (node.primCount != 1) {
                pdf *= 1.f / node.primCount;
            }
            break;
        }

        LightBVH leftNode = bvh[node.left];
        LightBVH rightNode = bvh[node.right];

        float leftImp = leftNode.flux / pow(owl::length(leftNode.aabbMid - si.p), 2.f);
        float rightImp = rightNode.flux / pow(owl::length(rightNode.aabbMid - si.p), 2.f);
        float sum = leftImp + rightImp;

        leftImp = leftImp / sum;
        rightImp = rightImp / sum;

        if (point.x >= leftNode.aabbMin.x && point.x <= leftNode.aabbMax.x
            && point.y >= leftNode.aabbMin.y && point.y <= leftNode.aabbMax.y
            && point.z >= leftNode.aabbMin.z && point.z <= leftNode.aabbMax.z) {
            nodeIdx = node.left;
            pdf *= leftImp;
        }
        else if (point.x >= rightNode.aabbMin.x && point.x <= rightNode.aabbMax.x
            && point.y >= rightNode.aabbMin.y && point.y <= rightNode.aabbMax.y
            && point.z >= rightNode.aabbMin.z && point.z <= rightNode.aabbMax.z) {
            nodeIdx = node.right;
            pdf *= rightImp;
        }
        else {
            break;
        }
    }

    return pdf;
}

__device__
void stochasticTraverseLBVH(LightBVH* bvh, int bvhHeight, int rootNodeIdx, SurfaceInteraction& si, int& selectedIdx,
                    float& lightSelectionPdf, vec2f randVec)
{
    selectedIdx = -1;
    lightSelectionPdf = 1.f;

    float r1 = randVec.x;
    float r2 = randVec.y;

    int nodeIdx = rootNodeIdx;
    for (int i = 0; i < bvhHeight + 1; i++) {
        LightBVH node = bvh[nodeIdx];

        // If leaf
        if (node.left == 0 && node.right == 0) {
            if (node.primCount == 1) {
                selectedIdx = node.primIdx;
            }
            else {
                selectedIdx = node.primIdx + round(r1 * (node.primCount-1));
                lightSelectionPdf *= 1.f / node.primCount;
            }

            break;
        }

        LightBVH leftNode = bvh[node.left];
        LightBVH rightNode = bvh[node.right];

        float leftImp = leftNode.flux / pow(owl::length(leftNode.aabbMid - si.p), 2.f);
        float rightImp = rightNode.flux / pow(owl::length(rightNode.aabbMid - si.p), 2.f);
        float sum = leftImp + rightImp;

        leftImp = leftImp / sum;
        rightImp = rightImp / sum;

        float eps = r2;
        if (eps < leftImp) {
            nodeIdx = node.left;
            lightSelectionPdf *= leftImp;
        }
        else {
            nodeIdx = node.right;
            lightSelectionPdf *= rightImp;
        }

        if (r1 < leftImp)
            r1 = r1 / leftImp;
        else
            r1 = (r1 - leftImp) / rightImp;

        if (r2 < leftImp)
            r2 = r2 / leftImp;
        else
            r2 = (r2 - leftImp) / rightImp;
    }
}

__device__ 
void selectFromLBVH(SurfaceInteraction& si, int& selectedIdx, float& lightSelectionPdf, vec2f rand0, vec2f rand1)
{
    // First, traverse the light TLAS and retrive the mesh light
    float lightTlasPdf = 1.f;
    int lightTlasIdx = 0;
    int lightTlasRootNodeIdx = 0;

    stochasticTraverseLBVH(optixLaunchParams.lightTlas, optixLaunchParams.lightTlasHeight, lightTlasRootNodeIdx,
        si, lightTlasIdx, lightTlasPdf, rand0);

    MeshLight meshLight = optixLaunchParams.meshLights[lightTlasIdx];

    // Finally, traverse the light BLAS and get the actual triangle
    float lightBlasPdf = 1.f;
    int lightBlasIdx = 0;
    int lightBlasRootNodeIdx = meshLight.bvhIdx;

    stochasticTraverseLBVH(optixLaunchParams.lightBlas, meshLight.bvhHeight, lightBlasRootNodeIdx,
        si, lightBlasIdx, lightBlasPdf, rand1);

    selectedIdx = lightBlasIdx;
    lightSelectionPdf = lightTlasPdf * lightBlasPdf;
}

__device__ 
float pdfFromLBVH(SurfaceInteraction& si, vec3f point)
{
    int meshIdx = 0;
    float tlasPdf = deterministicTraverseLBVH(optixLaunchParams.lightTlas, optixLaunchParams.lightTlasHeight, 
        0, si, point, meshIdx);
    
    MeshLight meshLight = optixLaunchParams.meshLights[meshIdx];
    int triIdx = 0;
    float blasPdf = deterministicTraverseLBVH(optixLaunchParams.lightBlas, meshLight.bvhHeight,
        meshLight.bvhIdx, si, point, triIdx);

    return tlasPdf * blasPdf;
}

__device__
vec3f integrateOverPolygon(SurfaceInteraction& si, vec3f ltc_mat[3], vec3f ltc_mat_inv[3], float amplitude,
    vec3f iso_frame[3], TriLight& triLight)
{
    vec3f lv1 = triLight.v1;
    vec3f lv2 = triLight.v2;
    vec3f lv3 = triLight.v3;
    vec3f lemit = triLight.emit;
    vec3f lnormal = triLight.normal;
    float larea = triLight.area;

    // Move to origin and normalize
    lv1 = owl::normalize(lv1 - si.p);
    lv2 = owl::normalize(lv2 - si.p);
    lv3 = owl::normalize(lv3 - si.p);

    vec3f cg = normalize(lv1 + lv2 + lv3);
    if (owl::dot(-cg, lnormal) < 0.f)
        return vec3f(0.f);

    lv1 = owl::normalize(apply_mat(si.to_local, lv1));
    lv2 = owl::normalize(apply_mat(si.to_local, lv2));
    lv3 = owl::normalize(apply_mat(si.to_local, lv3));

    lv1 = owl::normalize(apply_mat(iso_frame, lv1));
    lv2 = owl::normalize(apply_mat(iso_frame, lv2));
    lv3 = owl::normalize(apply_mat(iso_frame, lv3));

    float diffuse_shading = 0.f;
    float ggx_shading = 0.f;

    vec3f diff_clipped[5] = { lv1, lv2, lv3, lv1, lv1 };
    int diff_vcount = clipPolygon(3, diff_clipped);
    
    if (diff_vcount == 3) {
        diffuse_shading = integrateEdge(diff_clipped[0], diff_clipped[1]);
        diffuse_shading += integrateEdge(diff_clipped[1], diff_clipped[2]);
        diffuse_shading += integrateEdge(diff_clipped[2], diff_clipped[0]);
        diffuse_shading = owl::abs(diffuse_shading);
    }
    else if (diff_vcount == 4) {
        diffuse_shading = integrateEdge(diff_clipped[0], diff_clipped[1]);
        diffuse_shading += integrateEdge(diff_clipped[1], diff_clipped[2]);
        diffuse_shading += integrateEdge(diff_clipped[2], diff_clipped[3]);
        diffuse_shading += integrateEdge(diff_clipped[3], diff_clipped[0]);
        diffuse_shading = owl::abs(diffuse_shading);
    }

    diff_clipped[0] = owl::normalize(apply_mat(ltc_mat_inv, lv1));
    diff_clipped[1] = owl::normalize(apply_mat(ltc_mat_inv, lv2));
    diff_clipped[2] = owl::normalize(apply_mat(ltc_mat_inv, lv3));
    diff_clipped[3] = owl::normalize(apply_mat(ltc_mat_inv, lv1));
    diff_clipped[4] = owl::normalize(apply_mat(ltc_mat_inv, lv1));

    vec3f ltc_clipped[5] = { diff_clipped[0], diff_clipped[1], diff_clipped[2], diff_clipped[3], diff_clipped[4] };
    int ltc_vcount = clipPolygon(diff_vcount, ltc_clipped);

    if (ltc_vcount == 3) {
        ggx_shading = integrateEdge(ltc_clipped[0], ltc_clipped[1]);
        ggx_shading += integrateEdge(ltc_clipped[1], ltc_clipped[2]);
        ggx_shading += integrateEdge(ltc_clipped[2], ltc_clipped[0]);
        ggx_shading = owl::abs(ggx_shading);
    }
    else if (ltc_vcount == 4) {
        ggx_shading = integrateEdge(ltc_clipped[0], ltc_clipped[1]);
        ggx_shading += integrateEdge(ltc_clipped[1], ltc_clipped[2]);
        ggx_shading += integrateEdge(ltc_clipped[2], ltc_clipped[3]);
        ggx_shading += integrateEdge(ltc_clipped[3], ltc_clipped[0]);
        ggx_shading = owl::abs(ggx_shading);
    }
    else if (ltc_vcount == 5) {
        ggx_shading = integrateEdge(ltc_clipped[0], ltc_clipped[1]);
        ggx_shading += integrateEdge(ltc_clipped[1], ltc_clipped[2]);
        ggx_shading += integrateEdge(ltc_clipped[2], ltc_clipped[3]);
        ggx_shading += integrateEdge(ltc_clipped[3], ltc_clipped[4]);
        ggx_shading += integrateEdge(ltc_clipped[4], ltc_clipped[0]);
        ggx_shading = owl::abs(ggx_shading);
    }

    vec3f color = (si.diffuse * lemit * diffuse_shading) + (amplitude * lemit * ggx_shading);
    return color;
}

__device__ 
vec3f ltcDirectLightingLBVH(SurfaceInteraction& si, LCGRand& rng)
{
    vec3f normal_local(0.f, 0.f, 1.f);

    vec2f rand0(lcg_randomf(rng), lcg_randomf(rng));
    vec2f rand1(lcg_randomf(rng), lcg_randomf(rng));

    if (si.wo_local.z < 0.f)
        return vec3f(0.f);

    /* Analytic shading via LTCs */
    vec3f ltc_mat[3], ltc_mat_inv[3];
    float alpha = si.alpha;
    float theta = sphericalTheta(si.wo_local);

    float amplitude = 1.f;
    fetchLtcMat(alpha, theta, ltc_mat, amplitude);
    matrixInverse(ltc_mat, ltc_mat_inv);

    vec3f iso_frame[3];

    iso_frame[0] = si.wo_local;
    iso_frame[0].z = 0.f;
    iso_frame[0] = normalize(iso_frame[0]);
    iso_frame[2] = normal_local;
    iso_frame[1] = normalize(owl::cross(iso_frame[2], iso_frame[0]));

    int selectedIdx[MAX_LTC_LIGHTS * 2] = { -1 };
    int selectedEnd = 0;

    int ridx = 0;
    float rpdf = 0.f;
    selectFromLBVH(si, ridx, rpdf, rand0, rand1);

    selectedIdx[selectedEnd++] = ridx;

    for (int i = 0; i < MAX_LTC_LIGHTS*2; i++) {
        if (selectedEnd == optixLaunchParams.numTriLights)
            break;

        rand0 = vec2f(lcg_randomf(rng), lcg_randomf(rng));
        rand1 = vec2f(lcg_randomf(rng), lcg_randomf(rng));

        ridx = 0;
        rpdf = 0.f;
        selectFromLBVH(si, ridx, rpdf, rand0, rand1);

        bool found = false;
        for (int j = 0; j < selectedEnd; j++) {
            if (selectedIdx[j] == ridx) {
                found = true;
                break;
            }
        }

        if (!found) {
            selectedIdx[selectedEnd++] = ridx;
        }
    }

    vec3f color(0.f, 0.f, 0.f);
    for (int i = 0; i < selectedEnd; i++) {
        color += integrateOverPolygon(si, ltc_mat, ltc_mat_inv, amplitude, iso_frame,
            optixLaunchParams.triLights[selectedIdx[i]]);
    }

    return color;
}

// __device__ 
// vec3f ltcDirectLightingLBVH(SurfaceInteraction& si, bool useBst)
// {
//     const vec2i pixelId = owl::getLaunchIndex();
//     owl::common::LCG<MAX_LTC_LIGHTS*4> rng(pixelId.x * pixelId.y, optixLaunchParams.accumId);
//     float eps1 = rng();
//     float eps2 = rng();
// 
//     vec3f wo_local = normalize(apply_mat(si.to_local, si.wo));
//     if (wo_local.z < 0.f)
//         return vec3f(0.f);
// 
//     vec3f normal_local(0.f, 0.f, 1.f);
//     vec3f color(0.0, 0.0, 0.0);
// 
//     /* Analytic shading via LTCs */
//     vec3f ltc_mat[3], ltc_mat_inv[3];
//     float alpha = si.alpha;
//     float theta = sphericalTheta(wo_local);
// 
//     float amplitude = 1.f;
//     fetchLtcMat(alpha, theta, ltc_mat, amplitude);
//     matrixInverse(ltc_mat, ltc_mat_inv);
// 
//     vec3f iso_frame[3];
// 
//     iso_frame[0] = wo_local;
//     iso_frame[0].z = 0.f;
//     iso_frame[0] = normalize(iso_frame[0]);
//     iso_frame[2] = normal_local;
//     iso_frame[1] = normalize(owl::cross(iso_frame[2], iso_frame[0]));
// 
//     int selectedIdx[MAX_LTC_LIGHTS * 2] = { -1 };
//     int selectedEnd = 0;
// 
//     if (useBst) {
//         BST set[MAX_LTC_LIGHTS * 2];
//         int setEnd = 0;
// 
//         int numTriLights = optixLaunchParams.numTriLights;
// 
//         int ridx = -1;
//         float rpdf = 1.f;
//         traverseLBVH(si, ridx, rpdf, vec2f(rng(), rng()));
// 
//         set[setEnd++].data = ridx;
//         selectedIdx[selectedEnd++] = ridx;
// 
//         [[ unroll ]]
//         for (int i = 0; i < MAX_LTC_LIGHTS; i++) {
//             ridx = -1;
//             rpdf = 1.f;
//             traverseLBVH(si, ridx, rpdf, vec2f(rng(), rng()));
// 
//             int setIdx = 0;
//             bool found = false;
//             [[ unroll ]]
//             for (int j = 0; j < MAX_LTC_LIGHTS; j++) {
// 
//                 // If found
//                 if (set[setIdx].data == ridx) {
//                     found = true;
//                     break;
//                 }
// 
//                 // Insert if empty node
//                 if (set[setIdx].data == -1 && set[setIdx].left == -1 && set[setIdx].right == -1) {
//                     set[setIdx].data = ridx;
//                     break;
//                 }
// 
//                 // If child
//                 if (set[setIdx].data != -1 && set[setIdx].left == -1 && set[setIdx].right == -1) {
//                     set[setEnd++].data = ridx;
//                     set[setEnd++].data = -1;
// 
//                     if (ridx > set[setIdx].data) {
//                         set[setIdx].right = setEnd - 2;
//                         set[setIdx].left = setEnd - 1;
//                     }
//                     else {
//                         set[setIdx].right = setEnd - 1;
//                         set[setIdx].left = setEnd - 2;
//                     }
// 
//                     break;
//                 }
// 
//                 if (ridx > set[setIdx].data) {
//                     setIdx = set[setIdx].right;
//                 }
//                 else {
//                     setIdx = set[setIdx].left;
//                 }
// 
//             }
// 
//             if (!found)
//                 selectedIdx[selectedEnd++] = ridx;
//         }
//     }
//     else {
//         int numTriLights = optixLaunchParams.numTriLights;
// 
//         int ridx = -1;
//         float rpdf = 1.f;
//         traverseLBVH(si, ridx, rpdf, vec2f(rng(), rng()));
//         selectedIdx[selectedEnd++] = ridx;
// 
//         for (int i = 0; i < MAX_LTC_LIGHTS*2; i++) {
//             traverseLBVH(si, ridx, rpdf, vec2f(rng(), rng()));
// 
//             bool found = false;
//             for (int j = 0; j < selectedEnd; j++) {
//                 if (selectedIdx[j] == ridx) {
//                     found = true;
//                     break;
//                 }
//             }
// 
//             if (!found) {
//                 selectedIdx[selectedEnd++] = ridx;
//             }
//         }
//     }
// 
//     [[ unroll ]]
//     for (int i = 0; i < selectedEnd; i++) {
//         color += integrateOverPolygon(si, ltc_mat, ltc_mat_inv, amplitude, iso_frame,
//             optixLaunchParams.triLights[selectedIdx[i]]);
//     }
// 
//     return color;
// }

__device__
vec3f ltcDirecLighingBaseline(SurfaceInteraction& si, LCGRand& rng)
{
    vec3f wo_local = normalize(apply_mat(si.to_local, si.wo));
    if (wo_local.z < 0.f)
        return vec3f(0.f);

    vec3f normal_local(0.f, 0.f, 1.f);
    vec3f color(0.0, 0.0, 0.0);

    /* Analytic shading via LTCs */
    vec3f ltc_mat[3], ltc_mat_inv[3];
    float alpha = si.alpha;
    float theta = sphericalTheta(wo_local);

    float amplitude = 1.f;
    fetchLtcMat(alpha, theta, ltc_mat, amplitude);
    matrixInverse(ltc_mat, ltc_mat_inv);

    vec3f iso_frame[3];

    iso_frame[0] = wo_local;
    iso_frame[0].z = 0.f;
    iso_frame[0] = normalize(iso_frame[0]);
    iso_frame[2] = normal_local;
    iso_frame[1] = normalize(owl::cross(iso_frame[2], iso_frame[0]));

    for (int lidx = 0; lidx < optixLaunchParams.numTriLights; lidx++) {
        color += integrateOverPolygon(si, ltc_mat, ltc_mat_inv, amplitude, iso_frame, 
                                            optixLaunchParams.triLights[lidx]);
    }

    return color;
}

__device__
vec3f estimateDirectLightingLBVH(SurfaceInteraction& si, LCGRand& rng, int type)
{
    vec2f rand0(lcg_randomf(rng), lcg_randomf(rng));
    vec2f rand1(lcg_randomf(rng), lcg_randomf(rng));
    vec2f rand2(lcg_randomf(rng), lcg_randomf(rng));
    vec2f rand3(lcg_randomf(rng), lcg_randomf(rng));

    int selectedTriLight = 0;
    float lightSelectionPdf = 0.f;
    selectFromLBVH(si, selectedTriLight, lightSelectionPdf, rand0, rand1);

    vec3f lightSample = vec3f(0.f);
    vec3f brdfSample = vec3f(0.f);

    if (type == 0) {
        lightSample = sampleLightSource(si, selectedTriLight, lightSelectionPdf, rand2, false);
    }
    else if (type == 1) {
        brdfSample = sampleBRDF(si, lightSelectionPdf, rand3, false);
    }
    else if (type == 2) {
        brdfSample = sampleBRDF(si, lightSelectionPdf, rand2, true);
        lightSample = sampleLightSource(si, selectedTriLight, lightSelectionPdf, rand3, true);
    }

    // Make sure there are no negative colors!
    vec3f color = lightSample + brdfSample;
    color.x = owl::max(0.f, color.x);
    color.y = owl::max(0.f, color.y);
    color.z = owl::max(0.f, color.z);

    return color;
}

__device__
vec3f estimateDirectLighting(SurfaceInteraction& si, LCGRand& rng, int type)
{
    vec2f rand1 = vec2f(lcg_randomf(rng), lcg_randomf(rng));
    vec2f rand2 = vec2f(lcg_randomf(rng), lcg_randomf(rng));

    int selectedTriLight = round(lcg_randomf(rng) * (optixLaunchParams.numTriLights-1));
    float lightSelectionPdf = 1.f / optixLaunchParams.numTriLights;

    vec3f lightSample = vec3f(0.f);
    vec3f brdfSample = vec3f(0.f);

    if (type == 0) {
        lightSample = sampleLightSource(si, selectedTriLight, lightSelectionPdf, rand1, false);
    }
    else if (type == 1) {
        brdfSample = sampleBRDF(si, lightSelectionPdf, rand2, false);
    }
    else if (type == 2) {
        brdfSample = sampleBRDF(si, lightSelectionPdf, rand1, true);
        lightSample = sampleLightSource(si, selectedTriLight, lightSelectionPdf, rand2, true);
    }

    // Make sure there are no negative colors!
    vec3f color = lightSample + brdfSample;
    color.x = owl::max(0.f, color.x);
    color.y = owl::max(0.f, color.y);
    color.z = owl::max(0.f, color.z);

    return color;
}

__device__
vec3f sampleLightSource(SurfaceInteraction si, int lightIdx, float lightSelectionPdf, vec2f rand, bool mis)
{
    vec3f color(0.f, 0.f, 0.f);
    float light_pdf = 0.f, brdf_pdf = 0.f;
    TriLight triLight = optixLaunchParams.triLights[lightIdx];

    vec3f lv1 = triLight.v1;
    vec3f lv2 = triLight.v2;
    vec3f lv3 = triLight.v3;
    vec3f lnormal = triLight.normal;
    vec3f lemit = triLight.emit;
    float larea = triLight.area;

    vec3f lpoint = samplePointOnTriangle(lv1, lv2, lv3, rand.x, rand.y);
    si.wi = normalize(lpoint - si.p);
    si.wi_local = normalize(apply_mat(si.to_local, si.wi));

    float xmy = pow(owl::length(lpoint - si.p), 2.f);
    float lDotWi = owl::abs(owl::dot(lnormal, -si.wi));

    light_pdf = lightSelectionPdf * (xmy / (larea * lDotWi));

    ShadowRay ray;
    ray.origin = si.p + 1e-3f * si.n_geom;
    ray.direction = si.wi;

    ShadowRayData srd;
    owl::traceRay(optixLaunchParams.world, ray, srd);

    if (si.wo_local.z > 0.f && si.wi_local.z > 0.f && srd.visibility != vec3f(0.f) && light_pdf > 0.f && owl::dot(-si.wi, lnormal) > 0.f) {
        vec3f brdf = evaluate_brdf(si.wo_local, si.wi_local, si.diffuse, si.alpha);
        brdf_pdf = get_brdf_pdf(si.alpha, si.wo_local, normalize(si.wo_local + si.wi_local));

        if (mis && brdf_pdf > 0.f) {
            float weight = PowerHeuristic(1, light_pdf, 1, brdf_pdf);
            color += brdf * lemit * owl::abs(si.wi_local.z) * weight / light_pdf;
        }
        else if(!mis) {
            color += brdf * lemit * owl::abs(si.wi_local.z) / light_pdf;
        }
    }

    return color;
}

__device__
vec3f sampleBRDF(SurfaceInteraction si, float lightSelectionPdf, vec2f rand, bool mis)
{
    si.wi_local = sample_GGX(rand, si.alpha, si.wo_local);
    si.wi = normalize(apply_mat(si.to_world, si.wi_local));

    ShadowRay ray;
    ShadowRayData srd;
    ray.origin = si.p + 1e-3f * si.n_geom;
    ray.direction = si.wi;
    owl::traceRay(optixLaunchParams.world, ray, srd);

    vec3f color(0.f, 0.f, 0.f);
    float light_pdf = 0.f, brdf_pdf = 0.f;

    if (si.wi_local.z > 0.f && si.wo_local.z > 0.f && srd.visibility != vec3f(0.f)) {
        float xmy = pow(owl::length(srd.point - si.p), 2.f);
        float lDotWi = owl::abs(owl::dot(srd.normal, -si.wi));
        light_pdf = lightSelectionPdf * (xmy / (srd.area * lDotWi));

        vec3f brdf = evaluate_brdf(si.wo_local, si.wi_local, si.diffuse, si.alpha);
        brdf_pdf = get_brdf_pdf(si.alpha, si.wo_local, normalize(si.wo_local + si.wi_local));

        if (mis && light_pdf > 0.f && brdf_pdf > 0.f) {
            float weight = PowerHeuristic(1, brdf_pdf, 1, light_pdf);
            color += brdf * srd.emit * owl::abs(si.wi_local.z) * weight / brdf_pdf;
        }
        else if (!mis && brdf_pdf > 0.f) {
            color += brdf * srd.emit * owl::abs(si.wi_local.z) / brdf_pdf;
        }
    }

    return color;
}

