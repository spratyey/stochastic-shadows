#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2019-2020 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "deviceCode.h"
#include <owl/common/math/random.h>
#include <optix_device.h>

typedef owl::common::LCG<4> Random;

OPTIX_RAYGEN_PROGRAM(simpleRayGen)()
{
  const RayGenData &self = owl::getProgramData<RayGenData>();
  const vec2i pixelID = owl::getLaunchIndex();
  if (pixelID == owl::vec2i(0)) {
    printf("%sHello OptiX From your First RayGen Program%s\n",
           OWL_TERMINAL_CYAN,
           OWL_TERMINAL_DEFAULT);
  }

  Random rng(pixelID);
  
  const vec2f screen = (vec2f(pixelID)+vec2f(.5f)) / vec2f(self.fbSize);
  owl::Ray ray;
  ray.origin    
    = self.camera.pos;
  ray.direction 
    = normalize(self.camera.dir_00
                + screen.u * self.camera.dir_du
                + screen.v * self.camera.dir_dv);
  
  vec3f avgColor = 0.f;
  const int numSPP = 16;
  for (int i=0;i<numSPP;i++) {
    ray.time = rng();
    vec3f color;
    owl::traceRay(/*accel to trace against*/self.world,
                  /*the ray to trace*/ray,
                  /*prd*/color);
    
    avgColor += color;
  }
  avgColor *= 1.f/numSPP;
  const int fbOfs = pixelID.x+self.fbSize.x*pixelID.y;
    self.fbPtr[fbOfs]
      = owl::make_rgba(avgColor);
  }
  
inline __device__ vec3f lerp(const vec3f &A, const vec3f &B, float t)
{ return (1.f-t)*A + t*B; }
  
OPTIX_CLOSEST_HIT_PROGRAM(TriangleMesh)()
{
  vec3f &prd = owl::getPRD<vec3f>();

  const TrianglesGeomData &self = owl::getProgramData<TrianglesGeomData>();
  
  // compute normal:
  const int   primID  = optixGetPrimitiveIndex();
  const float time    = optixGetRayTime();
  const vec3i index   = self.index[primID];
  const vec3f &A0     = self.vertex0[index.x];
  const vec3f &B0     = self.vertex0[index.y];
  const vec3f &C0     = self.vertex0[index.z];
  const vec3f &A1     = self.vertex1[index.x];
  const vec3f &B1     = self.vertex1[index.y];
  const vec3f &C1     = self.vertex1[index.z];
  const vec3f A       = lerp(A0,A1,time);
  const vec3f B       = lerp(B0,B1,time);
  const vec3f C       = lerp(C0,C1,time);
  const vec3f Ng      = normalize(cross(B-A,C-A));

  const vec3f rayDir  = optixGetWorldRayDirection();
  prd = (.2f + .8f*fabs(dot(rayDir,Ng)))*self.color;
}

OPTIX_MISS_PROGRAM(miss)()
{
  const vec2i pixelID = owl::getLaunchIndex();

  const MissProgData &self = owl::getProgramData<MissProgData>();
  
  vec3f &prd = owl::getPRD<vec3f>();
  int pattern = (pixelID.x / 8) ^ (pixelID.y/8);
  prd = (pattern&1) ? self.color1 : self.color0;
}

